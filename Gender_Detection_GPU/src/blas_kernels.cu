#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"
#include <assert.h>

extern "C" {
#include "blas.h"
#include "hip/hip_runtime.h"
#include "utils.h"
}


__global__ void scale_bias_kernel(float *output, float *biases, int n, int size)
{
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int filter = blockIdx.y;
    int batch = blockIdx.z;

    if(offset < size) output[(batch*n+filter)*size + offset] *= biases[filter];
}

void scale_bias_gpu(float *output, float *biases, int batch, int n, int size)
{
    dim3 dimGrid((size-1)/BLOCK + 1, n, batch);
    dim3 dimBlock(BLOCK, 1, 1);

    scale_bias_kernel<<<dimGrid, dimBlock>>>(output, biases, n, size);
    check_error(hipPeekAtLastError());
}

__global__ void add_bias_kernel(float *output, float *biases, int batch, int n, int size)
{
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= n*size*batch) return;
    int i = index % size;
    index /= size;
    int j = index % n;
    index /= n;
    int k = index;

    output[(k*n+j)*size + i] += biases[j];
}

void add_bias_gpu(float *output, float *biases, int batch, int n, int size)
{
    int num = n*size*batch;

    add_bias_kernel<<<cuda_gridsize(num), BLOCK>>>(output, biases, batch, n, size);
    check_error(hipPeekAtLastError());
}

__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
	int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if (index >= N) return;
	int f = (index / spatial) % filters;

	x[index] = (x[index] - mean[f]) / (sqrtf(variance[f] + .00001f));
}


extern "C" void normalize_gpu(float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    size_t N = batch*filters*spatial;
    normalize_kernel<<<cuda_gridsize(N), BLOCK>>>(N, x, mean, variance, batch, filters, spatial);
    check_error(hipPeekAtLastError());
}

extern "C" void copy_gpu(int N, float * X, int INCX, float * Y, int INCY)
{
    copy_gpu_offset(N, X, 0, INCX, Y, 0, INCY);
}

__global__ void copy_kernel(int N, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
	int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if (i < N) Y[i*INCY + OFFY] = X[i*INCX + OFFX];
}

extern "C" void copy_gpu_offset(int N, float * X, int OFFX, int INCX, float * Y, int OFFY, int INCY)
{
    copy_kernel<<<cuda_gridsize(N), BLOCK>>>(N, X, OFFX, INCX, Y, OFFY, INCY);
    check_error(hipPeekAtLastError());
}

extern "C" void axpy_gpu(int N, float ALPHA, float * X, int INCX, float * Y, int INCY)
{
	axpy_gpu_offset(N, ALPHA, X, 0, INCX, Y, 0, INCY);
}

__global__ void axpy_kernel(int N, float ALPHA, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
	int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if (i < N) Y[OFFY + i * INCY] += ALPHA * X[OFFX + i * INCX];
}

extern "C" void axpy_gpu_offset(int N, float ALPHA, float * X, int OFFX, int INCX, float * Y, int OFFY, int INCY)
{
	axpy_kernel << <cuda_gridsize(N), BLOCK >> > (N, ALPHA, X, OFFX, INCX, Y, OFFY, INCY);
	check_error(hipPeekAtLastError());
}

__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
	int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if (i < N) Y[i*INCY] = pow(X[i*INCX], ALPHA);
}

extern "C" void pow_gpu(int N, float ALPHA, float * X, int INCX, float * Y, int INCY)
{
	pow_kernel << <cuda_gridsize(N), BLOCK >> > (N, ALPHA, X, INCX, Y, INCY);
	check_error(hipPeekAtLastError());
}

__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
	int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if (i < N) Y[i*INCY] *= X[i*INCX];
}

extern "C" void mul_gpu(int N, float * X, int INCX, float * Y, int INCY)
{
	mul_kernel << <cuda_gridsize(N), BLOCK >> > (N, X, INCX, Y, INCY);
	check_error(hipPeekAtLastError());
}

__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
	int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if (i < N) X[i*INCX] = ALPHA;
}

extern "C" void fill_gpu(int N, float ALPHA, float * X, int INCX)
{
	fill_kernel << <cuda_gridsize(N), BLOCK >> > (N, ALPHA, X, INCX);
	check_error(hipPeekAtLastError());
}

__global__ void const_kernel(int N, float ALPHA, float *X, int INCX)
{
	int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if (i < N) X[i*INCX] = ALPHA;
}

extern "C" void const_gpu(int N, float ALPHA, float * X, int INCX)
{
	const_kernel << <cuda_gridsize(N), BLOCK >> > (N, ALPHA, X, INCX);
	check_error(hipPeekAtLastError());
}

__global__ void shortcut_kernel(int size, int minw, int minh, int minc, int stride, int sample, int batch, int w1, int h1, int c1, float *add, int w2, int h2, int c2, float s1, float s2, float *out)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= size) return;
    int i = id % minw;
    id /= minw;
    int j = id % minh;
    id /= minh;
    int k = id % minc;
    id /= minc;
    int b = id % batch;

    int out_index = i*sample + w2*(j*sample + h2*(k + c2*b));
    int add_index = i*stride + w1*(j*stride + h1*(k + c1*b));
    out[out_index] = s1*out[out_index] + s2*add[add_index];
    //out[out_index] += add[add_index];
}

extern "C" void shortcut_gpu(int batch, int w1, int h1, int c1, float *add, int w2, int h2, int c2, float s1, float s2, float *out)
{
    int minw = (w1 < w2) ? w1 : w2;
    int minh = (h1 < h2) ? h1 : h2;
    int minc = (c1 < c2) ? c1 : c2;

    int stride = w1/w2;
    int sample = w2/w1;
    assert(stride == h1/h2);
    assert(sample == h2/h1);
    if(stride < 1) stride = 1;
    if(sample < 1) sample = 1;

    int size = batch * minw * minh * minc;
    shortcut_kernel<<<cuda_gridsize(size), BLOCK>>>(size, minw, minh, minc, stride, sample, batch, w1, h1, c1, add, w2, h2, c2, s1, s2, out);
    check_error(hipPeekAtLastError());
}

__global__ void scal_kernel(int N, float ALPHA, float *X, int INCX)
{
	int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if (i < N) X[i*INCX] *= ALPHA;
}

extern "C" void scal_gpu(int N, float ALPHA, float * X, int INCX)
{
	scal_kernel << <cuda_gridsize(N), BLOCK >> > (N, ALPHA, X, INCX);
	check_error(hipPeekAtLastError());
}

__global__ void add_kernel(int N, float ALPHA, float *X, int INCX)
{
	int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if (i < N) X[i*INCX] += ALPHA;
}

extern "C" void add_gpu(int N, float ALPHA, float * X, int INCX)
{
	add_kernel << <cuda_gridsize(N), BLOCK >> > (N, ALPHA, X, INCX);
	check_error(hipPeekAtLastError());
}

__global__ void smooth_l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
	int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if (i < n) {
		float diff = truth[i] - pred[i];
		float abs_val = fabsf(diff);
		if (abs_val < 1) {
			error[i] = diff * diff;
			delta[i] = diff;
		}
		else {
			error[i] = 2 * abs_val - 1;
			delta[i] = (diff > 0) ? 1 : -1;
		}
	}
}

extern "C" void smooth_l1_gpu(int n, float *pred, float *truth, float *delta, float *error)
{
	smooth_l1_kernel << <cuda_gridsize(n), BLOCK >> > (n, pred, truth, delta, error);
	check_error(hipPeekAtLastError());
}

__global__ void l2_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
	int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if (i < n) {
		float diff = truth[i] - pred[i];
		error[i] = diff * diff; //I know this is technically wrong, deal with it.
		delta[i] = diff;
	}
}

extern "C" void l2_gpu(int n, float *pred, float *truth, float *delta, float *error)
{
	l2_kernel << <cuda_gridsize(n), BLOCK >> > (n, pred, truth, delta, error);
	check_error(hipPeekAtLastError());
}

__global__ void l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
	int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if (i < n) {
		float diff = truth[i] - pred[i];
		error[i] = abs(diff);
		delta[i] = (diff > 0) ? 1 : -1;
	}
}

extern "C" void l1_gpu(int n, float *pred, float *truth, float *delta, float *error)
{
	l1_kernel << <cuda_gridsize(n), BLOCK >> > (n, pred, truth, delta, error);
	check_error(hipPeekAtLastError());
}

__global__ void wgan_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
	int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if (i < n) {
		error[i] = truth[i] ? -pred[i] : pred[i];
		delta[i] = (truth[i] > 0) ? 1 : -1;
	}
}

extern "C" void wgan_gpu(int n, float *pred, float *truth, float *delta, float *error)
{
	wgan_kernel << <cuda_gridsize(n), BLOCK >> > (n, pred, truth, delta, error);
	check_error(hipPeekAtLastError());
}

__global__ void supp_kernel(int N, float ALPHA, float *X, int INCX)
{
	int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if (i < N) {
		if ((X[i*INCX] * X[i*INCX]) < (ALPHA * ALPHA)) X[i*INCX] = 0;
	}
}

extern "C" void supp_gpu(int N, float ALPHA, float * X, int INCX)
{
	supp_kernel << <cuda_gridsize(N), BLOCK >> > (N, ALPHA, X, INCX);
	check_error(hipPeekAtLastError());
}

__global__ void mask_kernel(int n, float *x, float mask_num, float *mask, float val)
{
	int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if (i < n && mask[i] == mask_num) x[i] = val;
}

extern "C" void mask_gpu(int N, float * X, float mask_num, float * mask, float val)
{
	mask_kernel << <cuda_gridsize(N), BLOCK >> > (N, X, mask_num, mask, val);
	check_error(hipPeekAtLastError());
}

__global__ void scale_mask_kernel(int n, float *x, float mask_num, float *mask, float scale)
{
	int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if (i < n && mask[i] == mask_num) x[i] *= scale;
}

extern "C" void scale_mask_gpu(int N, float * X, float mask_num, float * mask, float scale)
{
	scale_mask_kernel << <cuda_gridsize(N), BLOCK >> > (N, X, mask_num, mask, scale);
	check_error(hipPeekAtLastError());
}

__global__ void softmax_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
	int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if (i < n) {
		float t = truth[i];
		float p = pred[i];
		error[i] = (t) ? -log(p) : 0;
		delta[i] = t - p;
	}
}

extern "C" void softmax_x_ent_gpu(int n, float *pred, float *truth, float *delta, float *error)
{
	softmax_x_ent_kernel << <cuda_gridsize(n), BLOCK >> > (n, pred, truth, delta, error);
	check_error(hipPeekAtLastError());
}

__global__ void logistic_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
	int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if (i < n) {
		float t = truth[i];
		float p = pred[i];
		error[i] = -t * log(p + .0000001) - (1 - t)*log(1 - p + .0000001);
		delta[i] = t - p;
	}
}

extern "C" void logistic_x_ent_gpu(int n, float *pred, float *truth, float *delta, float *error)
{
	logistic_x_ent_kernel << <cuda_gridsize(n), BLOCK >> > (n, pred, truth, delta, error);
	check_error(hipPeekAtLastError());
}

__global__ void flatten_kernel(int N, float *x, int spatial, int layers, int batch, int forward, float *out)
{
	int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if (i >= N) return;
	int in_s = i % spatial;
	i = i / spatial;
	int in_c = i % layers;
	i = i / layers;
	int b = i;

	int i1 = b * layers*spatial + in_c * spatial + in_s;
	int i2 = b * layers*spatial + in_s * layers + in_c;

	if (forward) out[i2] = x[i1];
	else out[i1] = x[i2];
}

extern "C" void flatten_gpu(float *x, int spatial, int layers, int batch, int forward, float *out)
{
	int size = spatial * batch*layers;
	flatten_kernel << <cuda_gridsize(size), BLOCK >> > (size, x, spatial, layers, batch, forward, out);
	check_error(hipPeekAtLastError());
}

__global__ void reorg_kernel(int N, float *x, int w, int h, int c, int batch, int stride, int forward, float *out)
{
	int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if (i >= N) return;
	int in_index = i;
	int in_w = i % w;
	i = i / w;
	int in_h = i % h;
	i = i / h;
	int in_c = i % c;
	i = i / c;
	int b = i % batch;

	int out_c = c / (stride*stride);

	int c2 = in_c % out_c;
	int offset = in_c / out_c;
	int w2 = in_w * stride + offset % stride;
	int h2 = in_h * stride + offset / stride;
	//printf("%d\n", offset);
	int out_index = w2 + w * stride*(h2 + h * stride*(c2 + out_c * b));

	// printf("%d %d %d\n", w2, h2, c2);
	 //printf("%d %d\n", in_index, out_index);
	 //if(out_index >= N || out_index < 0) printf("bad bad bad \n");

	if (forward) out[out_index] = x[in_index];
	else out[in_index] = x[out_index];
	//if(forward) out[1] = x[1];
	//else out[0] = x[0];
}

extern "C" void reorg_gpu(float *x, int w, int h, int c, int batch, int stride, int forward, float *out)
{
	int size = w * h*c*batch;
	reorg_kernel << <cuda_gridsize(size), BLOCK >> > (size, x, w, h, c, batch, stride, forward, out);
	check_error(hipPeekAtLastError());
}

__global__ void l2norm_kernel(int N, float *x, float *dx, int batch, int filters, int spatial)
{
	int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if (index >= N) return;
	int b = index / spatial;
	int i = index % spatial;
	int f;
	float sum = 0;
	for (f = 0; f < filters; ++f) {
		int index = b * filters*spatial + f * spatial + i;
		sum += powf(x[index], 2);
	}
	sum = sqrtf(sum);
	if (sum == 0) sum = 1;
	//printf("%f\n", sum);
	for (f = 0; f < filters; ++f) {
		int index = b * filters*spatial + f * spatial + i;
		x[index] /= sum;
		dx[index] = (1 - x[index]) / sum;
	}
}

extern "C" void l2normalize_gpu(float *x, float *dx, int batch, int filters, int spatial)
{
	size_t N = batch * spatial;
	l2norm_kernel << <cuda_gridsize(N), BLOCK >> > (N, x, dx, batch, filters, spatial);
	check_error(hipPeekAtLastError());
}

__global__ void weighted_sum_kernel(int n, float *a, float *b, float *s, float *c)
{
	int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if (i < n) {
		c[i] = s[i] * a[i] + (1 - s[i])*(b ? b[i] : 0);
	}
}

extern "C" void weighted_sum_gpu(float *a, float *b, float *s, int num, float *c)
{
	weighted_sum_kernel << <cuda_gridsize(num), BLOCK >> > (num, a, b, s, c);
	check_error(hipPeekAtLastError());
}

__device__ void softmax_device(float *input, int n, float temp, int stride, float *output)
{
	int i;
	float sum = 0;
	float largest = -INFINITY;
	for (i = 0; i < n; ++i) {
		int val = input[i*stride];
		largest = (val > largest) ? val : largest;
	}
	for (i = 0; i < n; ++i) {
		float e = expf(input[i*stride] / temp - largest / temp);
		sum += e;
		output[i*stride] = e;
	}
	for (i = 0; i < n; ++i) {
		output[i*stride] /= sum;
	}
}

__global__ void softmax_kernel(float *input, int n, int batch, int batch_offset, int groups, int group_offset, int stride, float temp, float *output)
{
	int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if (id >= batch * groups) return;
	int b = id / groups;
	int g = id % groups;
	softmax_device(input + b * batch_offset + g * group_offset, n, temp, stride, output + b * batch_offset + g * group_offset);
}

extern "C" void softmax_gpu(float *input, int n, int batch, int batch_offset, int groups, int group_offset, int stride, float temp, float *output)
{
	softmax_kernel << <cuda_gridsize(batch*groups), BLOCK >> > (input, n, batch, batch_offset, groups, group_offset, stride, temp, output);
	check_error(hipPeekAtLastError());
}

__global__ void softmax_tree_kernel(float *input, int spatial, int batch, int stride, float temp, float *output, int groups, int *group_size, int *group_offset)
{
	int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if (id >= spatial * batch*groups) return;
	int s = id % spatial;
	id = id / spatial;
	int g = id % groups;
	int b = id / groups;
	int goff = group_offset[g] * spatial;
	int boff = b * stride;
	softmax_device(input + goff + boff + s, group_size[g], temp, spatial, output + goff + boff + s);
}

extern "C" void softmax_tree(float *input, int spatial, int batch, int stride, float temp, float *output, tree hier)
{
	int *tree_groups_size = cuda_make_int_array(hier.group_size, hier.groups);
	int *tree_groups_offset = cuda_make_int_array(hier.group_offset, hier.groups);
	/*
	   static int *tree_groups_size = 0;
	   static int *tree_groups_offset = 0;
	   if(!tree_groups_size){
	   tree_groups_size = cuda_make_int_array(hier.group_size, hier.groups);
	   tree_groups_offset = cuda_make_int_array(hier.group_offset, hier.groups);
	   }
	 */
	int num = spatial * batch*hier.groups;
	softmax_tree_kernel << <cuda_gridsize(num), BLOCK >> > (input, spatial, batch, stride, temp, output, hier.groups, tree_groups_size, tree_groups_offset);
	check_error(hipPeekAtLastError());
	cuda_free((float *)tree_groups_size);
	cuda_free((float *)tree_groups_offset);
}

__global__ void upsample_kernel(size_t N, float *x, int w, int h, int c, int batch, int stride, int forward, float scale, float *out)
{
	size_t i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if (i >= N) return;
	int out_index = i;
	int out_w = i % (w*stride);
	i = i / (w*stride);
	int out_h = i % (h*stride);
	i = i / (h*stride);
	int out_c = i % c;
	i = i / c;
	int b = i % batch;

	int in_w = out_w / stride;
	int in_h = out_h / stride;
	int in_c = out_c;

	int in_index = b * w*h*c + in_c * w*h + in_h * w + in_w;


	if (forward) out[out_index] += scale * x[in_index];
	else atomicAdd(x + in_index, scale * out[out_index]);
}
extern "C" void upsample_gpu(float *in, int w, int h, int c, int batch, int stride, int forward, float scale, float *out)
{
	size_t size = w * h*c*batch*stride*stride;
	upsample_kernel << <cuda_gridsize(size), BLOCK >> > (size, in, w, h, c, batch, stride, forward, scale, out);
	check_error(hipPeekAtLastError());
}